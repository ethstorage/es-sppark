#include "hip/hip_runtime.h"
// Copyright Jiazheng Liu, EthStorage
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

// Some constant macro only be used in this file
#define MAX_THREAD_NUM 1024
// 8 is the row of gate constraint, because we extend the domain by 8
// NOTE: The caller must make sure the buffer is at length of domain_size + 8
//       Otherwise it would cause out of bound error
#define NEXT(index, domain_size) (index + 8)
#define ONE fr_t::one()
#define TWO (fr_t::one() + fr_t::one())
#define THREE (fr_t::one() + fr_t::one() + fr_t::one())
#define FOUR (fr_t::one() + fr_t::one() + fr_t::one() + fr_t::one())
#define NINE (THREE * THREE)
#define EIGHTEEN (NINE * TWO)
#define EIGHTY_ONE (NINE * NINE)
#define EIGHTY_THREE (EIGHTY_ONE + TWO)
#define K1 (THREE + FOUR)  // 7
#define K2 (THREE * FOUR + ONE)  // 13
#define K3 (TWO * K1 + THREE) // 17

// Hardcode the SBOX in constant time
#define POW_SBOX(n) ((n) * (n) * (n) * (n) * (n))
#define SQAURE(n) ((n) * (n))

// This MACRO is used to generate the argument list of following function
// And it would be used in following files as well
// All of them are equal length, except w_l, w_r, w_4, they are extended by 8
#define POINTER_LIST(X) \
    X(w_l)              \
    X(w_r)              \
    X(w_o)              \
    X(w_4)              \
    X(q_l)              \
    X(q_r)              \
    X(q_o)              \
    X(q_4)              \
    X(q_hl)             \
    X(q_hr)             \
    X(q_h4)             \
    X(q_c)              \
    X(q_arith)          \
    X(q_m)              \
    X(r_s)              \
    X(l_s)              \
    X(fbsm_s)           \
    X(vgca_s)           \
    X(pi)               \
    X(z)                \
    X(perm_linear)      \
    X(sigma_l)          \
    X(sigma_r)          \
    X(sigma_o)          \
    X(sigma_4)          \
    X(q_lookup)         \
    X(table)            \
    X(f)                \
    X(h1)               \
    X(h2)               \
    X(z2)               \
    X(l1)               \
    X(l1_alpha_sq)      \
    X(v_h_coset) 


// Auxilary list
// Challenges has 5 elements, curve_params has 2 elements
// Permutation parameters has 6 elements
#define AUX_LIST(X) \
    X(challenges)   \
    X(curve_params) \
    X(perm_params)

#define RANGE_CHALLENGE challenges[0]
#define LOGIC_CHALLENGE challenges[1]
#define FIXED_BASE_CHALLENGE challenges[2]
#define VAR_BASE_CHALLENGE challenges[3]
#define LOOKUP_CHALLENGE challenges[4]

#define P_COEFF_A curve_params[0]
#define P_COEFF_D curve_params[1]

#define ALPHA perm_params[0]
#define BETA perm_params[1]
#define GAMMA perm_params[2]
#define DELTA perm_params[3]
#define EPSILON perm_params[4]
#define ZETA perm_params[5]

// Compose a argument list of following function
#define MAKE_PTR_ARGUMENT(var) , const fr_t* var

// Compose a parameter list of following function
#define MAKE_PARAMETER(var) , var

// Total argument
#define TOTAL_ARGUMENT \
    POINTER_LIST(MAKE_PTR_ARGUMENT) AUX_LIST(MAKE_PTR_ARGUMENT)

#define TOTAL_PARAMETER \
    POINTER_LIST(MAKE_PARAMETER) AUX_LIST(MAKE_PARAMETER)

#define PRODUCT_POINTER_LIST(X) \
    X(root) \
    X(gate_sigma0) \
    X(gate_sigma1) \
    X(gate_sigma2) \
    X(gate_sigma3) \
    X(gate_wire0) \
    X(gate_wire1) \
    X(gate_wire2) \
    X(gate_wire3) \

#define PRODUCT_AUX_LIST(X) \
    X(ks) \
    X(beta) \
    X(gamma) \

#define PRODUCT_ARGUMENT \
    PRODUCT_POINTER_LIST(MAKE_PTR_ARGUMENT) PRODUCT_AUX_LIST(MAKE_PTR_ARGUMENT)

#define PRODUCT_PARAMETER \
    PRODUCT_POINTER_LIST(MAKE_PARAMETER) PRODUCT_AUX_LIST(MAKE_PARAMETER)

#define LOOKUP_PRODUCT_POINTER_LIST(X) \
    X(f) \
    X(t) \
    X(t_next) \
    X(h_1) \
    X(h_1_next) \
    X(h_2) \

#define LOOKUP_PRODUCT_AUX_LIST(X) \
    X(delta) \
    X(epsilon) \

#define LOOKUP_PRODUCT_ARGUMENT \
    LOOKUP_PRODUCT_POINTER_LIST(MAKE_PTR_ARGUMENT) LOOKUP_PRODUCT_AUX_LIST(MAKE_PTR_ARGUMENT)

#define LOOKUP_PRODUCT_PARAMETER \
    LOOKUP_PRODUCT_POINTER_LIST(MAKE_PARAMETER) LOOKUP_PRODUCT_AUX_LIST(MAKE_PARAMETER)

#define LINEAR_POLY_POINTER_LIST(X) \
    X(q_m) \
    X(q_l) \
    X(q_r) \
    X(q_o) \
    X(q_4) \
    X(q_hl) \
    X(q_hr) \
    X(q_h4) \
    X(q_c) \
    X(z_poly) \
    X(fourth_sigma) \
    X(t_1_poly) \
    X(t_2_poly) \
    X(t_3_poly) \
    X(t_4_poly) \
    X(t_5_poly) \
    X(t_6_poly) \
    X(t_7_poly) \
    X(t_8_poly) \

#define LINEAR_POLY_AUX_LIST(X) \
    X(wit_vals) \
    X(perm_vals) \

#define LINEAR_POLY_ARGUMENT \
    LINEAR_POLY_POINTER_LIST(MAKE_PTR_ARGUMENT) LINEAR_POLY_AUX_LIST(MAKE_PTR_ARGUMENT), const uint64_t* power 

#define LINEAR_POLY_PARAMETER \
    LINEAR_POLY_POINTER_LIST(MAKE_PARAMETER) LINEAR_POLY_AUX_LIST(MAKE_PARAMETER), power

/*-------------------------GATE SAT---------------------------------------*/
__device__ __forceinline__ fr_t compute_quotient_i(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    return ((w_l[i] * w_r[i] * q_m[i])
            + (w_l[i] * q_l[i])
            + (w_r[i] * q_r[i])
            + (w_o[i] * q_o[i])
            + (w_4[i] * q_4[i])
            + (POW_SBOX(w_l[i]) * q_hl[i])
            + (POW_SBOX(w_r[i]) * q_hr[i])
            + (POW_SBOX(w_4[i]) * q_h4[i])
            + q_c[i])
            * q_arith[i];
}

__device__ __forceinline__ fr_t delta(fr_t f)
{
    fr_t f_1 = f - ONE;
    fr_t f_2 = f - TWO;
    fr_t f_3 = f - THREE;
    return f * f_1 * f_2 * f_3; 
}

__device__ __forceinline__ fr_t range_quoteint_term(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
   fr_t kappa = RANGE_CHALLENGE * RANGE_CHALLENGE;
   fr_t kappa_sq = kappa * kappa;
   fr_t kappa_cu = kappa_sq * kappa;
   fr_t b1 = delta(w_o[i] - FOUR * w_4[i]);
   fr_t b2 = delta(w_r[i] - FOUR * w_o[i]) * kappa;
   fr_t b3 = delta(w_l[i] - FOUR * w_r[i]) * kappa_sq;
   // NOTICE: w_4 is next one, should add next line
   fr_t b4 = delta(w_4[NEXT(i, domain_size)] - FOUR * w_l[i]) * kappa_cu;

   return r_s[i] * (b1 + b2 + b3 + b4) * RANGE_CHALLENGE;
}

__device__ __forceinline__ fr_t delta_xor_and(fr_t a, fr_t b, fr_t w, fr_t c, fr_t q_c)
{
    fr_t F = w
        * (w * (FOUR * w - EIGHTEEN * (a + b) + EIGHTY_ONE)
            + EIGHTEEN * (SQAURE(a) + SQAURE(b))
            - EIGHTY_ONE * (a + b)
            + EIGHTY_THREE);
    fr_t E = THREE * (a + b + c) - (TWO * F);
    fr_t B = q_c * ((NINE * c) - THREE * (a + b));
    return E + B;
}

__device__ __forceinline__ fr_t logic_quotient_term(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    fr_t kappa = RANGE_CHALLENGE * RANGE_CHALLENGE;
    fr_t kappa_sq = kappa * kappa;
    fr_t kappa_cu = kappa_sq * kappa;
    fr_t kappa_qu = kappa_cu * kappa;

    fr_t a = w_l[NEXT(i, domain_size)] - FOUR * w_l[i];
    fr_t c_0 = delta(a);

    fr_t b = w_r[NEXT(i, domain_size)] - FOUR * w_r[i];
    fr_t c_1 = delta(b) * kappa;

    fr_t d = w_4[NEXT(i, domain_size)] - FOUR * w_4[i];
    fr_t c_2 = delta(d) * kappa_sq;

    fr_t w = w_o[i];
    fr_t c_3 = (w - a * b) * kappa_cu;

    fr_t c_4 = delta_xor_and(a, b, w, d, q_c[i]) * kappa_qu;

    return l_s[i] * (c_0 + c_1 + c_2 + c_3 + c_4) * LOGIC_CHALLENGE;
}

// Extracts the bit value from the accumulated bit.
__device__ __forceinline__ fr_t extract_bit(fr_t curr_acc, fr_t next_acc)
{
    return next_acc - curr_acc - curr_acc;
}

/// Ensures that the bit is either `+1`, `-1`, or `0`
__device__ __forceinline__ fr_t check_bit_consistency(fr_t bit)
{
    return bit * (bit - ONE) * (bit + ONE);
}

__device__ __forceinline__ fr_t fixed_base_quoteint_term(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    fr_t kappa = SQAURE(FIXED_BASE_CHALLENGE);
    fr_t kappa_sq = SQAURE(kappa);
    fr_t kappa_cu = kappa_sq * kappa;

    fr_t x_beta_eval = q_l[i];
    fr_t y_beta_eval = q_r[i];

    fr_t acc_x = w_l[i];
    fr_t acc_x_next = w_l[NEXT(i, domain_size)];
    fr_t acc_y = w_r[i];
    fr_t acc_y_next = w_r[NEXT(i, domain_size)];

    fr_t xy_alpha = w_o[i];

    fr_t accumulated_bit = w_4[i];
    fr_t accumulated_bit_next = w_4[NEXT(i, domain_size)];
    fr_t bit = extract_bit(accumulated_bit, accumulated_bit_next);

    // Check bit consistency
    fr_t bit_consistency = check_bit_consistency(bit);

    fr_t y_alpha = SQAURE(bit) * (y_beta_eval - ONE) + ONE;
    fr_t x_alpha = x_beta_eval * bit;

    // xy_alpha consistency check
    fr_t xy_consistency = ((bit * q_c[i]) - xy_alpha) * kappa;

    // x accumulator consistency check
    fr_t x_3 = acc_x_next;
    fr_t lhs = x_3 + (x_3 * xy_alpha * acc_x * acc_y * P_COEFF_D);
    fr_t rhs = (x_alpha * acc_y) + (y_alpha * acc_x);
    fr_t x_acc_consistency = (lhs - rhs) * kappa_sq;

    // y accumulator consistency check
    fr_t y_3 = acc_y_next;
    lhs = y_3 - (y_3 * xy_alpha * acc_x * acc_y * P_COEFF_D);
    rhs = y_alpha * acc_y - P_COEFF_A * x_alpha * acc_x;
    fr_t y_acc_consistency = (lhs - rhs) * kappa_cu;

    fr_t checks = bit_consistency
        + x_acc_consistency
        + y_acc_consistency
        + xy_consistency;

    return fbsm_s[i] * checks * FIXED_BASE_CHALLENGE;
}

__device__ __forceinline__ fr_t curve_addition_quotient_term(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    fr_t x_1 = w_l[i];
    fr_t x_3 = w_l[NEXT(i, domain_size)];
    fr_t y_1 = w_r[i];
    fr_t y_3 = w_r[NEXT(i, domain_size)];
    fr_t x_2 = w_o[i];
    fr_t y_2 = w_4[i];
    fr_t x1_y2 = w_4[NEXT(i, domain_size)];

    fr_t kappa = SQAURE(VAR_BASE_CHALLENGE);

    // Check that `x1 * y2` is correct
    fr_t xy_consistency = x_1 * y_2 - x1_y2;

    fr_t y1_x2 = y_1 * x_2;
    fr_t y1_y2 = y_1 * y_2;
    fr_t x1_x2 = x_1 * x_2;

    // Check that `x_3` is correct
    fr_t x3_lhs = x1_y2 + y1_x2;
    fr_t x3_rhs = x_3 + (x_3 * P_COEFF_D * x1_y2 * y1_x2);
    fr_t x3_consistency = (x3_lhs - x3_rhs) * kappa;

    // Check that `y_3` is correct
    fr_t y3_lhs = y1_y2 - P_COEFF_A * x1_x2;
    fr_t y3_rhs = y_3 - y_3 * P_COEFF_D * x1_y2 * y1_x2;
    fr_t y3_consistency = (y3_lhs - y3_rhs) * SQAURE(kappa);

    return vgca_s[i] * (xy_consistency + x3_consistency + y3_consistency) * VAR_BASE_CHALLENGE;
}

__device__ __forceinline__ fr_t gate_sat_term(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    return compute_quotient_i(i, domain_size TOTAL_PARAMETER) + 
                range_quoteint_term(i, domain_size TOTAL_PARAMETER) +
                logic_quotient_term(i, domain_size TOTAL_PARAMETER) +
                fixed_base_quoteint_term(i, domain_size TOTAL_PARAMETER) +
                curve_addition_quotient_term(i, domain_size TOTAL_PARAMETER) +
                pi[i];
}

/*--------------------------------------PERMUTATION--------------------------------------------*/
__device__ __forceinline__ fr_t compute_quotient_identity_range_check_i(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    fr_t x = perm_linear[i];
    return (w_l[i] + BETA * x + GAMMA)
        * (w_r[i] + (BETA * K1 * x) + GAMMA)
        * (w_o[i] + (BETA * K2 * x) + GAMMA)
        * (w_4[i] + (BETA * K3 * x) + GAMMA)
        * z[i]
        * ALPHA;
}

__device__ __forceinline__ fr_t compute_quotient_copy_range_check_i(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    fr_t left_sigma_eval = sigma_l[i];
    fr_t right_sigma_eval = sigma_r[i];
    fr_t out_sigma_eval = sigma_o[i];
    fr_t fourth_sigma_eval = sigma_4[i];
    fr_t product = (w_l[i] + (BETA * left_sigma_eval) + GAMMA)
        * (w_r[i] + (BETA * right_sigma_eval) + GAMMA)
        * (w_o[i] + (BETA * out_sigma_eval) + GAMMA)
        * (w_4[i] + (BETA * fourth_sigma_eval) + GAMMA)
        * z[NEXT(i, domain_size)]
        * ALPHA;
    return product; 
}

__device__ __forceinline__ fr_t compute_quotient_term_check_one_i(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    return (z[i] - ONE) * l1_alpha_sq[i];
}

__device__ __forceinline__ fr_t permutation_term(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
    return compute_quotient_identity_range_check_i(i, domain_size TOTAL_PARAMETER)
           - compute_quotient_copy_range_check_i(i, domain_size TOTAL_PARAMETER)
           + compute_quotient_term_check_one_i(i, domain_size TOTAL_PARAMETER);
}

/*---------------------------------------LOOKUP-------------------------------------------------*/
// Linear combination of a series of values
__device__ __forceinline__ fr_t lc(const fr_t* coeffs, fr_t x, size_t n)
{
    // Horner's method
    fr_t acc = coeffs[n];
    for (size_t i = 1; i < n; ++i) {
        acc = acc * x + coeffs[n - i];
    }
    return acc;
}

__device__ __forceinline__ fr_t lookup_term(size_t i, size_t domain_size TOTAL_ARGUMENT)
{
   
    fr_t lookup_sep_sq = SQAURE(LOOKUP_CHALLENGE);
    fr_t lookup_sep_cu = lookup_sep_sq * LOOKUP_CHALLENGE;
    fr_t one_plus_delta = DELTA + ONE;
    fr_t epsilon_one_plus_delta = EPSILON * one_plus_delta;

    // q_lookup(X) * (a(X) + zeta * b(X) + (zeta^2 * c(X)) + (zeta^3 * d(X)
    // - f(X))) * α_1
    fr_t wit[4] = {w_l[i], w_r[i], w_o[i], w_4[i]};
    fr_t compressed_tuple = lc(wit, ZETA, 4);
    fr_t a = q_lookup[i] * (compressed_tuple - f[i]) * LOOKUP_CHALLENGE;

    // z2(X) * (1+δ) * (ε+f(X)) * (ε*(1+δ) + t(X) + δt(Xω)) * lookup_sep^2
    fr_t b_0 = EPSILON + f[i];
    fr_t b_1 = epsilon_one_plus_delta + table[i] + DELTA * table[NEXT(i, domain_size)];
    fr_t b = z2[i] * one_plus_delta * b_0 * b_1 * lookup_sep_sq;

    // − z2(Xω) * (ε*(1+δ) + h1(X) + δ*h2(X)) * (ε*(1+δ) + h2(X) + δ*h1(Xω))
    // * lookup_sep^2
    fr_t c_0 = epsilon_one_plus_delta + h1[i] + DELTA * h2[i];
    fr_t c_1 = epsilon_one_plus_delta + h2[i] + DELTA * h1[NEXT(i, domain_size)];
    fr_t c = z2[NEXT(i, domain_size)] * c_0 * c_1 * lookup_sep_sq;

    fr_t d = (z2[i] - ONE) * l1[i] * lookup_sep_cu;

    return a + b - c + d;
}

/*--------------------------------------PRODUCT ARGUMENT--------------------------------------------*/
__device__ __forceinline__ fr_t product_argment(size_t i, size_t domain_size PRODUCT_ARGUMENT) {
    fr_t _beta = beta[0];
    fr_t _gamma = gamma[0];
    fr_t numerator_product = (gate_wire0[i] + _beta * ks[0] * root[i] + _gamma)
        * (gate_wire1[i] + _beta * ks[1] * root[i] + _gamma) 
        * (gate_wire2[i] + _beta * ks[2] * root[i] + _gamma)
        * (gate_wire3[i] + _beta * ks[3] * root[i] + _gamma); 
    fr_t denominator_product = (gate_wire0[i] + _beta * gate_sigma0[i] + _gamma)
        * (gate_wire1[i] + _beta * gate_sigma1[i] + _gamma)
        * (gate_wire2[i] + _beta * gate_sigma2[i] + _gamma)
        * (gate_wire3[i] + _beta * gate_sigma3[i] + _gamma);
    return numerator_product / denominator_product;
}

/*--------------------------------------LOOKUP PRODUCT ARGUMENT--------------------------------------------*/
__device__ __forceinline__ fr_t lookup_product_argment(size_t i, size_t domain_size LOOKUP_PRODUCT_ARGUMENT) {
    fr_t _epsilon = epsilon[0];
    fr_t _delta = delta[0];
    fr_t one_plus_delta = _delta + ONE;
    fr_t epsilon_one_plus_delta = _epsilon * one_plus_delta;
    fr_t part_1 = one_plus_delta * (_epsilon + f[i]) 
        * (epsilon_one_plus_delta + t[i] + _delta * t_next[i]);
    fr_t part_2 = ((epsilon_one_plus_delta + h_1[i] + _delta * h_2[i])
        * (epsilon_one_plus_delta + h_2[i] + _delta * h_1_next[i])).reciprocal();
    return part_1 * part_2;
}

/*--------------------------------------LINEAR POLY: linear_poly_arithmetic--------------------------------------------*/
__device__ __forceinline__ fr_t linear_poly_arithmetic(size_t i, size_t domain_size LINEAR_POLY_ARGUMENT) {
    fr_t a_eval = wit_vals[0];
    fr_t b_eval = wit_vals[1];
    fr_t c_eval = wit_vals[2];
    fr_t d_eval = wit_vals[3];
    fr_t q_arith_eval = wit_vals[4];
    fr_t result = (
        q_m[i] * a_eval * b_eval
        + q_l[i] * a_eval 
        + q_r[i] * b_eval
        + q_o[i] * c_eval
        + q_4[i] * d_eval
        + q_hl[i] * POW_SBOX(a_eval)
        + q_hr[i] * POW_SBOX(b_eval)
        + q_h4[i] * POW_SBOX(d_eval)
        + q_c[i]
    ) * q_arith_eval;
    return result;
}

/*--------------------------------------LINEAR POLY: compute_lineariser_identity_range_check---------------------------------*/
__device__ __forceinline__ fr_t compute_lineariser_identity_range_check(size_t i, size_t domain_size LINEAR_POLY_ARGUMENT) {
    fr_t a_eval = wit_vals[0];
    fr_t b_eval = wit_vals[1];
    fr_t c_eval = wit_vals[2];
    fr_t d_eval = wit_vals[3];
    fr_t z_challenge = perm_vals[1];
    fr_t alpha = perm_vals[2];
    fr_t beta = perm_vals[3];
    fr_t gamma = perm_vals[4];

    fr_t beta_z = beta * z_challenge;
    // a_eval + beta * z_challenge + gamma
    fr_t a_0 = a_eval + beta_z;
    a_0 += gamma;

    // b_eval + beta * K1 * z_challenge + gamma
    fr_t beta_z_k1 = K1 * beta_z;
    fr_t a_1 = b_eval + beta_z_k1;
    a_1 += gamma;

    // c_eval + beta * K2 * z_challenge + gamma
    fr_t beta_z_k2 = K2 * beta_z;
    fr_t a_2 = c_eval + beta_z_k2;
    a_2 += gamma;

    // d_eval + beta * K3 * z_challenge + gamma
    fr_t beta_z_k3 = K3 * beta_z;
    fr_t a_3 = d_eval + beta_z_k3;
    a_3 += gamma;

    fr_t a = a_0 * a_1;
    a *= a_2;
    a *= a_3;
    a *= alpha; 

    return z_poly[i] * a;
}

/*--------------------------------------LINEAR POLY: compute_lineariser_copy_range_check---------------------------------*/
__device__ __forceinline__ fr_t compute_lineariser_copy_range_check(size_t i, size_t domain_size LINEAR_POLY_ARGUMENT) {
    fr_t a_eval = wit_vals[0];
    fr_t b_eval = wit_vals[1];
    fr_t c_eval = wit_vals[2];
    fr_t alpha = perm_vals[2];
    fr_t beta = perm_vals[3];
    fr_t gamma = perm_vals[4];
    fr_t sigma_1_eval = perm_vals[5];
    fr_t sigma_2_eval = perm_vals[6];
    fr_t sigma_3_eval = perm_vals[7];
    fr_t z_eval = perm_vals[8];

    // a_eval + beta * sigma_1 + gamma
    fr_t beta_sigma_1 = beta * sigma_1_eval;
    fr_t a_0 = a_eval + beta_sigma_1;
    a_0 += gamma;

    // b_eval + beta * sigma_2 + gamma
    fr_t beta_sigma_2 = beta * sigma_2_eval;
    fr_t a_1 = b_eval + beta_sigma_2;
    a_1 += gamma;

    // c_eval + beta * sigma_3 + gamma
    fr_t beta_sigma_3 = beta * sigma_3_eval;
    fr_t a_2 = c_eval + beta_sigma_3;
    a_2 += gamma;

    fr_t beta_z_eval = beta * z_eval;

    fr_t a = a_0 * a_1 * a_2;
    a *= beta_z_eval;
    a *= alpha; // (a_eval + beta * sigma_1 + gamma)(b_eval + beta * sigma_2 +
                // gamma)(c_eval + beta * sigma_3 + gamma) * beta * z_eval * alpha
    fr_t result = fourth_sigma[i] * a;
    // to negate a Fr
    return result.cneg(true);
}

/*--------------------------------------LINEAR POLY: compute_lineariser_check_is_one---------------------------------*/
__device__ __forceinline__ fr_t compute_lineariser_check_is_one(size_t i, size_t domain_size LINEAR_POLY_ARGUMENT) {
    fr_t a_eval = wit_vals[0];
    fr_t z_challenge = perm_vals[1];
    fr_t alpha = perm_vals[2];
    fr_t beta = perm_vals[3];
    fr_t gamma = perm_vals[4];
    fr_t sigma_1_eval = perm_vals[5];
    fr_t alpha_sq = SQAURE(alpha);

    // a_eval + beta * sigma_1 + gamma
    fr_t beta_sigma_1 = beta * sigma_1_eval;
    fr_t a_0 = a_eval + beta_sigma_1;
    a_0 += gamma;

    fr_t m = perm_vals[0]; 
    fr_t h = ONE;
    fr_t v_0_inv = m;
    const uint64_t p = power[0];
    // There is no explicit arithmetic precedence, so a brack must used for ^
    fr_t l_1_z = ((z_challenge^p) - h) /v_0_inv / (z_challenge - h);
    return z_poly[i] * (l_1_z * alpha_sq);
}

/*--------------------------------------LINEAR POLY: compute_quotient_tem---------------------------------*/
__device__ __forceinline__ fr_t compute_quotient_tem(size_t i, size_t domain_size LINEAR_POLY_ARGUMENT) {
    fr_t z_challenge_to_n = perm_vals[9];
    fr_t vanishing_poly_eval = perm_vals[10];
    fr_t z_2 = SQAURE(z_challenge_to_n);
    fr_t z_3 = z_2 * z_challenge_to_n;
    fr_t z_4 = z_3 * z_challenge_to_n;
    fr_t z_5 = z_4 * z_challenge_to_n;
    fr_t z_6 = z_5 * z_challenge_to_n;
    fr_t z_7 = z_6 * z_challenge_to_n;
    fr_t result = (t_1_poly[i] 
        + t_2_poly[i]*z_challenge_to_n 
        + t_3_poly[i]*z_2 
        + t_4_poly[i]*z_3
        + t_5_poly[i]*z_4
        + t_6_poly[i]*z_5
        + t_7_poly[i]*z_6
        + t_8_poly[i]*z_7
        ) * vanishing_poly_eval;
    // to negate a Fr
    return result.cneg(true);
}


/*----------------------------------FINAL KERNEL FUNCTION---------------------------------------*/
__launch_bounds__(MAX_THREAD_NUM, 1) __global__
void quotient_poly_kernel(const size_t domain_size, fr_t* out
                                TOTAL_ARGUMENT)
{
#if (__CUDACC_VER_MAJOR__-0) >= 11
    __builtin_assume(domain_size <= (1 << MAX_LG_DOMAIN_SIZE));
#endif
    const index_t tid = threadIdx.x + blockDim.x * (index_t)blockIdx.x;

    // out of range, just return
    if (tid > domain_size) {
        return;
    }

    fr_t numerator =   gate_sat_term(tid, domain_size TOTAL_PARAMETER)
                     + permutation_term(tid, domain_size TOTAL_PARAMETER)
                     + lookup_term(tid, domain_size TOTAL_PARAMETER);
    out[tid] = numerator / v_h_coset[tid];
    
}

__launch_bounds__(MAX_THREAD_NUM, 1) __global__
void product_argment_kernel(const uint lg_domain_size, fr_t* out
                                PRODUCT_ARGUMENT)
{
#if (__CUDACC_VER_MAJOR__-0) >= 11
    __builtin_assume(lg_domain_size <= MAX_LG_DOMAIN_SIZE);
#endif
    uint domain_size = 1 << lg_domain_size;
    const index_t tid = threadIdx.x + blockDim.x * (index_t)blockIdx.x;

    // out of range, just return
    if (tid > domain_size) {
        return;
    }

    out[tid] =  product_argment(tid, domain_size PRODUCT_PARAMETER);
}

__launch_bounds__(MAX_THREAD_NUM, 1) __global__
void lookup_product_argment_kernel(const uint lg_domain_size, fr_t* out
                                LOOKUP_PRODUCT_ARGUMENT)
{
#if (__CUDACC_VER_MAJOR__-0) >= 11
    __builtin_assume(lg_domain_size <= MAX_LG_DOMAIN_SIZE);
#endif
    uint domain_size = 1 << lg_domain_size;
    const index_t tid = threadIdx.x + blockDim.x * (index_t)blockIdx.x;

    // out of range, just return
    if (tid > domain_size) {
        return;
    }

    out[tid] =  lookup_product_argment(tid, domain_size LOOKUP_PRODUCT_PARAMETER);
}

__launch_bounds__(MAX_THREAD_NUM, 1) __global__
void linear_poly_kernel(const uint lg_domain_size, fr_t* out
                                LINEAR_POLY_ARGUMENT)
{
#if (__CUDACC_VER_MAJOR__-0) >= 11
    __builtin_assume(lg_domain_size <= MAX_LG_DOMAIN_SIZE);
#endif
    uint domain_size = 1 << lg_domain_size;
    const index_t tid = threadIdx.x + blockDim.x * (index_t)blockIdx.x;

    // out of range, just return
    if (tid > domain_size) {
        return;
    }
    out[tid] = linear_poly_arithmetic(tid, domain_size LINEAR_POLY_PARAMETER)
        + compute_lineariser_identity_range_check(tid, domain_size LINEAR_POLY_PARAMETER)
        + compute_lineariser_copy_range_check(tid, domain_size LINEAR_POLY_PARAMETER)
        + compute_lineariser_check_is_one(tid, domain_size LINEAR_POLY_PARAMETER)
        + compute_quotient_tem(tid, domain_size LINEAR_POLY_PARAMETER);
}

#undef MAX_THREAD_NUM
#undef NEXT
#undef ONE
#undef TWO
#undef THREE
#undef FOUR
#undef NINE
#undef EIGHTEEN
#undef EIGHTY_ONE
#undef EIGHTY_THREE
#undef K1
#undef K2
#undef K3
#undef POW_SBOX
#undef SQAURE
#undef RANGE_CHALLENGE
#undef LOGIC_CHALLENGE
#undef FIXED_BASE_CHALLENGE
#undef VAR_BASE_CHALLENGE
#undef LOOKUP_CHALLENGE
#undef P_COEFF_A
#undef P_COEFF_D
#undef ALPHA
#undef BETA
#undef GAMMA
#undef DELTA
#undef EPSILON
#undef ZETA