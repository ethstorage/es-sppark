// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381.hpp>
#elif defined(FEATURE_BLS12_377)
# include <ff/bls12-377.hpp>
#elif defined(FEATURE_PALLAS)
# include <ff/pasta.hpp>
#elif defined(FEATURE_VESTA)
# include <ff/pasta.hpp>
#else
# error "no FEATURE"
#endif

#include <ntt/ntt.cuh>
#include <ntt/arithmetic.cuh>

#ifndef __CUDA_ARCH__

extern "C"
RustError compute_ntt(size_t device_id, fr_t* inout, uint32_t lg_domain_size,
                      NTT::InputOutputOrder ntt_order,
                      NTT::Direction ntt_direction,
                      NTT::Type ntt_type)
{
    auto& gpu = select_gpu(device_id);

    return NTT::Base(gpu, inout, lg_domain_size,
                     ntt_order, ntt_direction, ntt_type);
}

extern "C"
RustError compute_gate_constraint(size_t device_id, uint32_t lg_domain_size,
                                  fr_t* out POINTER_LIST(MAKE_ARGUMENT))
{
    auto& gpu = select_gpu(device_id);

    return ARITHMETIC::gate_constraint(gpu, lg_domain_size, out POINTER_LIST(MAKE_PARAMETER));
}
#endif
