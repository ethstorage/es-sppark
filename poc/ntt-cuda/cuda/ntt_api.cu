// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>

#if defined(FEATURE_BLS12_381)
# include <ff/bls12-381.hpp>
#elif defined(FEATURE_BLS12_377)
# include <ff/bls12-377.hpp>
#elif defined(FEATURE_PALLAS)
# include <ff/pasta.hpp>
#elif defined(FEATURE_VESTA)
# include <ff/pasta.hpp>
#else
# error "no FEATURE"
#endif

#include <ntt/ntt.cuh>
#include <ntt/arithmetic.cuh>

#ifndef __CUDA_ARCH__

extern "C"
RustError compute_ntt(size_t device_id, fr_t* inout, uint32_t lg_domain_size,
                      NTT::InputOutputOrder ntt_order,
                      NTT::Direction ntt_direction,
                      NTT::Type ntt_type)
{
    auto& gpu = select_gpu(device_id);

    return NTT::Base(gpu, inout, lg_domain_size,
                     ntt_order, ntt_direction, ntt_type);
}

extern "C"
RustError compute_quotient_term(size_t device_id, uint32_t lg_domain_size,
                                  fr_t* out TOTAL_ARGUMENT)
{
    auto& gpu = select_gpu(device_id);

    return ARITHMETIC::quotient_poly_gpu(gpu, lg_domain_size, out TOTAL_PARAMETER);
}
#endif
